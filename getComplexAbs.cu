#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuda_funcs.h"

/* Need to try two different methods: 1) calculating PSD ahead of time and copying it to GPU, or 
   calculating it each time on the GPU
*/
__global__ void getComplexAbs(float *out, hipfftComplex *in, 
							int NX, int NY, int NZ) {
	
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int k = threadIdx.z + blockIdx.z*blockDim.z;
	int index = k*NX*NY+j*NX+i;

	if (i < NX && j < NY && k < NZ) {
		
		out[index] = sqrt(powf(in[index].x,2) + powf(in[index].y,2));
	}
}

void getComplexAbs(float *out, hipfftComplex *in, dataSize size) {

	dim3 dimGrid (int((size.x-0.5)/BSZ) + 1, int((size.y-0.5)/BSZ) + 1, size.z);
	dim3 dimBlock (BSZ, BSZ, 1);
	// Need to make complex numbers here
	getComplexAbs<<<dimGrid, dimBlock>>>(out, in, size.x, size.y, size.z);

}