#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hiprand.h>
#include <math.h>
#include <stdio.h>
#include "cuda_funcs.h"
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"

#define PI 3.14159265358979323846

//512 x 512 x 1000 in 32-bit floats => 1.05GB => 2.1GB "complex"
// my device has 1GB of memory, roughly 512 x 512 x 1000 x 32 bit
// will try 250 at a time first
// 1024 threads per block (warp size is 32)
int main() {

	float D = 2.0;
	float r0 = 0.1;
	float L0 = 100;
	//float l0 = 0.01;

	dataSize size;   //Might want to set up constructor and volume elem
	size.x = 512;
	size.y = 512;
	size.z = 1;

	char out_window[] = "Result";
	float out[size.x*size.y*size.z];

	float delta = D/size.x;

	hiprandGenerator_t gen;

	hipfftHandle plan;
	hipfftComplex *data, *shift_out;
	float *real_data;
	float *imag_data;
	CUDA_CALL(hipMalloc((void**)&data, sizeof(hipfftComplex)*size.x*size.y*size.z));
	CUDA_CALL(hipMalloc((void**)&real_data, sizeof(float)*size.x*size.y*size.z));
	CUDA_CALL(hipMalloc((void**)&imag_data, sizeof(float)*size.x*size.y*size.z));

	// Initialize the gpu "arrays" with randn numbers
	CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	/* Set seed */
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

	/* Generate real and imag normally random distributed numbers */
	CURAND_CALL(hiprandGenerateNormal(gen, real_data, size.x*size.y*size.z, 0.0, 1.0));
	CURAND_CALL(hiprandGenerateNormal(gen, imag_data, size.x*size.y*size.z, 0.0, 1.0));

	// Need to make complex numbers here
	makeComplexPSD(real_data, imag_data, data, r0, delta, L0, size);
	CUDA_CALL(hipFree(real_data));
	CUDA_CALL(hipFree(imag_data));
	CUDA_CALL(hipMalloc((void**)&shift_out, sizeof(hipfftComplex)*size.x*size.y*size.z));
	fftshift(shift_out, data, size.x, size.z);

	//2^a x 3^b is most efficient size
	/* Create a 2D plan */
	//Need advanced data layout to do batch in 2D using hipfftPlanMany
	hipfftPlan2d(&plan, size.x, size.y, HIPFFT_C2C);
	hipfftExecC2C(plan, shift_out, shift_out, HIPFFT_BACKWARD);
	
	fftshift(data, shift_out, size.x, size.z);
	CUDA_CALL(hipFree(shift_out));
	CUDA_CALL(hipMalloc((void**)&real_data, sizeof(float)*size.x*size.y*size.z));
	getComplexAbs(real_data, data, size);

	CUDA_CALL(hipMemcpy(out, real_data, size.x*size.y*size.z*sizeof(float), hipMemcpyDeviceToHost));

	cv::Mat screen = cv::Mat(size.x*size.z, size.y, CV_32FC1, &out);
    cv::imshow(out_window, screen);
    cv::waitKey(0);

    double min, max;
	cv::minMaxLoc(screen, &min, &max);
	printf("Min: %f\nMax: %f \n", min, max);

	cv::destroyAllWindows();

	/* Destroy the CUFFT plan */
	hipfftDestroy(plan);
	hipFree(data);
	hipFree(real_data);

}